//#include <cuda_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>

/// Kernel
__global__ void KernelTemplate(size_t elementsNo)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < elementsNo; i+=stride) {
        printf("index : %d\n", i);
    }

}

/// Main function
__host__ void CUDAfunctionTemplate(size_t elementsNo)
{
    // Send current data to GPU
//    sendDataToGPU(camPoseArray, cloudSize);

    // Kernel launch parameters
    int blockSize = 256;
    int numBlocks = (elementsNo + blockSize - 1) / blockSize;

    // Launch kernel
    KernelTemplate<<<numBlocks, blockSize>>>(elementsNo);

    // Wait for all kernels to finish
    hipDeviceSynchronize();

    // Get point cloud data from GPU
//    getDataFromGPU(cloudSize);
}

/// Kernel
__global__ void dotProductKernel(const float* vect1, const float* vect2, float *partialSum, size_t elementsNo)
{
    extern __shared__ float sdata[];

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // printf("dupae3 %f\n", vect1[0]);
    float temp = 0.0f;
    // printf("index : %d, value1: %f, value2: %f\n", index, vect1[index], vect2[index]);
    for (int idx = index; idx < elementsNo; idx+=stride) {
        temp += vect1[idx] * vect2[idx];
        // printf("index : %d, value1: %f, value2: %f\n", idx, vect1[idx], vect2[idx]);
    }

    int cacheIdx = threadIdx.x;
    sdata[cacheIdx] = temp;

    __syncthreads();  // Synchronize before reduction

    // Reduction within block
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIdx < i) {
            sdata[cacheIdx] += sdata[cacheIdx + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Write result from each block to global memory
    if (cacheIdx == 0) {
        // printf("cache id: %d, cache val %f\n", blockIdx.x, sdata[0]);
        partialSum[blockIdx.x] = sdata[0];
    }

}

/// Main function
__host__ void CUDAdotProduct(const float* vect1, const float* vect2, size_t elementsNo, float* result)
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    // Kernel launch parameters
    int threadsPerBlock = 256;
    // int numberOfBlocks = (elementsNo + threadsPerBlock - 1) / threadsPerBlock;
    int numBlocks = 32 * numberOfSMs;

    // Send current data to GPU
    float *vect1_d, *vect2_d, *d_partialSum;
    hipMalloc(&vect1_d, elementsNo * sizeof(float));
    hipMalloc(&vect2_d, elementsNo * sizeof(float));

    hipMemcpy(vect1_d, vect1, elementsNo * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vect2_d, vect2, elementsNo * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_partialSum, numBlocks * sizeof(float));

    printf("dupae1 %f\n", vect1[0]);
    // Launch kernel
    printf("num blocks %d, blocksize %d\n", numBlocks, threadsPerBlock);
    dotProductKernel<<<numBlocks, threadsPerBlock>>>(vect1_d, vect2_d, d_partialSum, elementsNo);
    printf("dupae2 %f\n", vect2[0]);

    hipError_t addVectorsErr;
    hipError_t asyncErr;

    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    // Wait for all kernels to finish
    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));


    // Copy partial sums back to CPU
    float* h_partialSum = new float[numBlocks];
    hipMemcpy(h_partialSum, d_partialSum, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    printf("partial sum[0] %f\n", h_partialSum[0]);

    // Final CPU reduction
    *result = 0;
    for (int i = 0; i < numBlocks; ++i)
        *result += h_partialSum[i];

    // Get point cloud data from GPU
    //    getDataFromGPU(cloudSize);
}
